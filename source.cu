#include "hip/hip_runtime.h"
%%cu

#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
#include <sys/time.h>
#define bias -0.3
#define n 1024
#define IM 60000
#define lim 60000
#define mx_layer 120

#define layer_loc "/content/drive/MyDrive/data/neuron1024/"
#define category_loc "/content/drive/MyDrive/data/"
#define feature_loc "/content/drive/MyDrive/data/"

using namespace std;

__global__ void CSR_SpMV(int *cm,int *row,float *val,float *d_Y,int* all_zero) {
    int ind = blockIdx.x*blockDim.x + threadIdx.x;
    
    __shared__ float Y[n];
    Y[ind]  = d_Y[ind];

    __syncthreads();

    float sum=0.0;
    for(int i=cm[ind];i<cm[ind+1];i++){
        //printf("%d\n",ptr);
        sum+=Y[row[i]]*val[i]*1.0;
    }
    __syncthreads();
    
    if(Y[ind]>=0.0)
      Y[ind]=sum+bias;

    //ReLU operation
    if(Y[ind]<0.0)Y[ind]=0.0;
    else if(Y[ind]>32.0)Y[ind]=32.0;

    if(Y[ind] != 0 && *all_zero == 1){
        *all_zero = 0;
    }
    d_Y[ind]  = Y[ind];
    
}

class Layercls {
    public:
        int *cm;
        int *row;
        int nnz;
        float *val;
        void add_layer(vector<Layercls> &lvec,Layercls &ly){
            lvec.push_back(ly);
        }
};

void store_layers(int max_layer, vector<Layercls> &layers ){
    
    for(int lay_no=1;lay_no<=max_layer;lay_no++){
      float* M;
      M = (float*)calloc(n*n,sizeof(float));
      int *cm, *row;
      float *val;
      string fname(layer_loc);
      vector<string> fpara={"n",to_string(n), "-l", to_string(lay_no), ".tsv"};
      for(string x:fpara){
          fname.append(x);
      }
      ifstream file(fname);

      string line;
      int NNZ=0;
      while(getline(file,line)){
          stringstream ss(line);
          int r,c;
          float v;
          ss >> r >> c >> v;
          r--;c--;
          M[r*n+c]=v;
          NNZ++;
      }
      cm=(int*)malloc((n+1)*sizeof(int));
      row=(int*)malloc(NNZ*sizeof(int));
      val=(float*)malloc(NNZ*sizeof(float));

      cm[0]=0;
      int ptr=0,nnz_cnt=0;
      for(int j=0;j<n;j++){
          nnz_cnt=0;
          for(int i=0;i<n;i++){
              if(M[i*n+j]!=0.0){
                  //cout<<i<<" "<<j<<" "<<M[i*n+j]<<" "<<ptr<<endl;
                  val[ptr]=M[i*n+j];
                  row[ptr]=i;
                  nnz_cnt++;
                  ptr++;
              }
          }
          cm[j+1]=cm[j]+nnz_cnt;
      }
      Layercls l1;
      l1.cm=cm;
      l1.row=row;
      l1.val=val;
      l1.nnz=NNZ;
      l1.add_layer(layers,l1);
      delete M;
    }
    
}
void load_feature_vector(vector<float*> &all_feature_vec){
    string fname(feature_loc);
    fname.append("sparse-images-1024.tsv");
    ifstream file(fname);
    float *all;
    all=(float*)calloc(IM*n,sizeof(float));
    string line;
    while(getline(file,line)){
        stringstream ss(line);
        int r,c;
        float val;
        ss >> r >> c >> val;
        r--;c--;
        all[r*n+c]=val;
    }

    for(int i=0;i<IM;i++){
        float *img;
        img=(float*)calloc(n,sizeof(float));
        for(int j=0;j<n;j++){
            img[j]=all[i*n+j];
        }
        all_feature_vec.push_back(img);
    }
    delete all;
}

void load_true_category(vector<int> &true_cat){
    string fname(category_loc);
    fname.append("neuron1024-l"+to_string(mx_layer)+"-categories.tsv");
    ifstream file(fname);
    string line;
    while(getline(file,line)){
        stringstream ss(line);
        int cat;
        ss >> cat;
        if(cat>lim) break;
        true_cat.push_back(cat);
    }
}

int main(){
    
    vector<Layercls> layers;
    store_layers(mx_layer,layers);

    vector<float*> all_img;
    load_feature_vector(all_img);
    

    int *d_cm, *d_row;
    
    hipMalloc(&d_cm,(n+1)*sizeof(int));
    hipMalloc(&d_row,n*n*sizeof(int));

    float *d_val;
    hipMalloc(&d_val,n*n*sizeof(float));
    

    float *Y;
    Y=(float*)malloc(n*sizeof(float));

    int *all_zero;
    int *d_all_zero;
    hipMalloc(&d_all_zero,sizeof(int));
    
    struct timeval begin, end;
    gettimeofday(&begin, 0);
    /* Timer Starts */

    vector<bool> is_cat(lim,true);

    vector<float*> d_img;
    for(int img=0;img<lim;img++){
        float* d_Y;
        hipMalloc(&d_Y,n*sizeof(float));
        hipMemcpy(d_Y,all_img[img],n*sizeof(float),hipMemcpyHostToDevice);
        d_img.push_back(d_Y);
    }

    for(int lay=0;lay<mx_layer;lay++){
        int NNZ=layers[lay].nnz;
        hipMemcpy(d_cm,layers[lay].cm,(n+1)*sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(d_row,layers[lay].row,NNZ*sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(d_val,layers[lay].val,NNZ*sizeof(float),hipMemcpyHostToDevice);
        
        for(int img=0;img<lim;img++){
          if(!is_cat[img])continue;
          *all_zero = 1;
          hipMemcpy(d_all_zero,all_zero,sizeof(int),hipMemcpyHostToDevice);
          CSR_SpMV<<<1,n>>> (d_cm,d_row,d_val,d_img[img],d_all_zero);
          hipMemcpy(all_zero,d_all_zero,sizeof(int),hipMemcpyDeviceToHost);
          if(*all_zero == 1) is_cat[img]=false;
        }
    }
    vector<int> cat;
    for(int i=0;i<lim;i++){
        if(is_cat[i]) cat.push_back(i+1);
    }
    /* Timer Stops */
    gettimeofday(&end, 0);
    long seconds = end.tv_sec - begin.tv_sec;
    long microseconds = end.tv_usec - begin.tv_usec;
    double elapsed = seconds + microseconds*1e-6;
    
    
    vector<int> true_cat;
    load_true_category(true_cat);

    // category comparison
    bool pas=(cat.size()==true_cat.size());
    for(int i=0;i<cat.size();i++){
        if(cat[i]!=true_cat[i]){
            pas=false;
            break;
        }
    }
    if(pas){
        float ssize=(lim*1.0/IM)*100;
        printf("Test Passed! \nSample size used: %f % \nTime measured: %.6f seconds.\n",ssize, elapsed);
    }
    else{
        printf("Test failed!!");
    }
    
    return 0;
}


